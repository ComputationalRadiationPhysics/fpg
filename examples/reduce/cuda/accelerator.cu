#include "hip/hip_runtime.h"
/* Copyright (c) 2018, Jan Stephan
 * All rights reserved.
 *
 * This software may be modified and distributed under the terms of the BSD
 * license. See the LICENSE file for details.
 */

#include <cstdint>
#include <cstdlib>
#include <cwchar>
#include <iostream>
#include <locale>
#include <string>
#include <vector>

#include "accelerator.h"

#define CHECK(cmd) \
{ \
        auto error = cmd; \
        if(error != hipSuccess) \
        { \
                    std::cerr << "Error: '" << hipGetErrorString(error) \
                              << "' (" << error << ") at " << __FILE__ << ":" \
                              << __LINE__  << std::endl; \
                    std::exit(EXIT_FAILURE); \
                } \
}

namespace acc
{
    namespace
    {
        auto cstream_ = hipStream_t{};
    }

    struct dev_ptr_impl
    {
        int* ptr;
        std::size_t size;

        ~dev_ptr_impl()
        {
            if(ptr != nullptr)
                CHECK(hipFree(ptr));
        }
    };

    dev_ptr::~dev_ptr()
    {
        if(p_impl != nullptr)
            delete p_impl;
    }

    struct dev_clock_impl
    {
        hipEvent_t event;
    };

    dev_clock::~dev_clock()
    {
        if(p_impl != nullptr)
            delete p_impl;
    }

    auto init() -> void
    {
        // set up default device
        auto dev_count = int{};
        CHECK(hipGetDeviceCount(&dev_count));

        std::cout << "Available accelerators: " << std::endl;
        for(auto d = 0; d < dev_count; ++d)
        {
            auto prop = hipDeviceProp_t{};
            CHECK(hipGetDeviceProperties(&prop, d));

            std::cout << "\t[" << d << "] " << prop.name << std::endl;
        }

        std::cout << std::endl;
        std::cout << "Select accelerator: ";
        auto index = 0;
        std::cin >> index;

        if(index >= dev_count)
        {
            std::cout << "I'm sorry, Dave. I'm afraid I can't do that."
                      << std::endl;
            std::exit(EXIT_FAILURE);
        }

        CHECK(hipSetDevice(index));
        CHECK(hipFree(nullptr)); // force context init

        CHECK(hipStreamCreate(&cstream_));
    }

    auto get_info() -> info
    {
        auto id = int{};
        CHECK(hipGetDevice(&id));

        auto prop = hipDeviceProp_t{};
        CHECK(hipGetDeviceProperties(&prop, id));

        auto name = std::string{prop.name};
        auto cc_major = prop.major;
        auto cc_minor = prop.minor;
        auto mem_clock = prop.memoryClockRate / 1000;
        auto clock = prop.clockRate / 1000;
        auto num_sm = prop.multiProcessorCount;
        return info{id, name, cc_major, cc_minor, mem_clock, clock, num_sm};
    }

    auto make_array(std::size_t size) -> dev_ptr
    {
        auto d_ptr = new dev_ptr_impl{nullptr, size};
        CHECK(hipMalloc(&(d_ptr->ptr), size * sizeof(int)));
        return dev_ptr{d_ptr};
    }

    auto copy_h2d(const std::vector<int>& src, dev_ptr& dst) -> void
    {
        CHECK(hipMemcpy(dst.p_impl->ptr, src.data(), src.size() * sizeof(int),
                        hipMemcpyHostToDevice));
    }

    auto copy_d2h(const dev_ptr& src, std::vector<int>& dst) -> void
    {
        CHECK(hipMemcpy(dst.data(), src.p_impl->ptr,
                        src.p_impl->size * sizeof(int),
                        hipMemcpyDeviceToHost));
    }

    __global__ void block_reduce(const int* data, int* result, std::size_t size)
    {
        __shared__ int scratch[1024]; 

        auto i = blockIdx.x * blockDim.x + threadIdx.x;

        if(i >= size)
            return;

        // avoid neutral element
        auto tsum = data[i];

        auto grid_size = gridDim.x * blockDim.x;
        i += grid_size;

        // GRID, read from global memory
        while((i + 3 * grid_size) < size)
        {
            tsum += data[i] + data[i + grid_size] + data[i + 2 * grid_size] +
                    data[i + 3 * grid_size];
            i += 4 * grid_size;
        }

        // tail
        while(i < size)
        {
            tsum += data[i];
            i += grid_size;
        }

        scratch[threadIdx.x] = tsum;
        __syncthreads();


        // BLOCK + WARP, read from shared memory
        #pragma unroll
        for(auto bs = blockDim.x, bsup = (blockDim.x + 1) / 2;
            bs > 1;
            bs /= 2, bsup = (bs + 1) / 2)
        {
            auto cond = threadIdx.x < bsup // first half of block
                        && (threadIdx.x + bsup) < blockDim.x
                        && (blockIdx.x * blockDim.x +
                            threadIdx.x + bsup) < size;

            if(cond)
            {
                scratch[threadIdx.x] += scratch[threadIdx.x + bsup];
            }
            __syncthreads();

            // store to global memory
            if(threadIdx.x == 0)
                result[blockIdx.x] = scratch[0];
        }
    }

    auto do_benchmark(const dev_ptr& data, dev_ptr& result, std::size_t size,
                      int blocks, int block_size) -> void
    {
        block_reduce<<<blocks, block_size, 0, cstream_>>>(data.p_impl->ptr,
                                                          result.p_impl->ptr,
                                                          size);
        
        block_reduce<<<1, block_size, 0, cstream_>>>(result.p_impl->ptr,
                                                     result.p_impl->ptr,
                                                     blocks);
    }

    auto start_clock() -> dev_clock
    {
        auto clock_impl = new dev_clock_impl{};
        CHECK(hipDeviceSynchronize());
        CHECK(hipEventCreate(&(clock_impl->event)));
        CHECK(hipEventRecord(clock_impl->event, cstream_));
        return dev_clock{clock_impl};
    }

    auto stop_clock() -> dev_clock
    {
        auto clock_impl = new dev_clock_impl{};
        CHECK(hipEventCreate(&(clock_impl->event)));
        CHECK(hipEventRecord(clock_impl->event, cstream_));
        CHECK(hipEventSynchronize(clock_impl->event));
        return dev_clock{clock_impl};
    }

    auto get_duration(const dev_clock& start, const dev_clock& stop) -> float
    {
        auto ms = float{};
        CHECK(hipGetLastError());
        CHECK(hipEventElapsedTime(&ms, start.p_impl->event,
                                   stop.p_impl->event));
        return ms;
    }
}

// vim:ft=cuda
