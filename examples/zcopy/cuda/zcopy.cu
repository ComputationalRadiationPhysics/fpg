
#include <hip/hip_runtime.h>
#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <limits>
#include <sstream>

#define CHECK(cmd) \
{ \
    auto error = cmd; \
    if(error != hipSuccess) \
    { \
        std::cerr << "Error: '" << hipGetErrorString(error) \
                  << "' (" << error << ") at " << __FILE__ << ":" \
                  << __LINE__ << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} \

__global__ void read_write(const float4* __restrict__ A,
                                 float4* __restrict__ B,
                           std::size_t elems)
{
    auto stride = gridDim.x * blockDim.x;
    for(auto i = blockIdx.x * blockDim.x + threadIdx.x; i < elems; i += stride)
    {
        B[i] = A[i];
    } 
}

__global__ void write(float4* __restrict__ B, std::size_t elems)
{
    auto stride = gridDim.x * blockDim.x;
    for(auto i = blockIdx.x * blockDim.x + threadIdx.x; i < elems; i += stride)
    {
        B[i] = make_float4(0.f, 0.f, 0.f, 0.f);
    }
}

auto do_benchmark(int sms, int max_blocks, std::ofstream& file,
                  int start_size, int stop_size) -> void
{
    constexpr auto iters = 10;
    constexpr auto max_mem = 1u << 31; // mem per vector
    constexpr auto max_elems = static_cast<int>(max_mem / sizeof(float4));

    for(auto block_size = start_size; block_size <= stop_size; block_size *= 2)
    {
        for(auto elems = block_size * sms; elems <= max_elems; elems *= 2)
        {
            // Allocate memory on device
            auto A_d = static_cast<float4*>(nullptr);
            auto B_d = static_cast<float4*>(nullptr);

            CHECK(hipMalloc(&A_d, sizeof(float4) * elems));
            CHECK(hipMalloc(&B_d, sizeof(float4) * elems));

            for(auto block_num = sms;
                     block_num <= std::min(elems / block_size, max_blocks);
                     block_num *= 2)
            {
                // Initialize device memory
                CHECK(hipMemset(A_d, 0x00, sizeof(float4) * elems)); // zero
                CHECK(hipMemset(B_d, 0xff, sizeof(float4) * elems)); // NaN

                auto mintime = std::numeric_limits<float>::max();
                for(auto k = 0; k < iters; ++k)
                {
                    auto start_event = hipEvent_t{};
                    auto stop_event = hipEvent_t{};
                    CHECK(hipDeviceSynchronize());
                    CHECK(hipEventCreate(&start_event));
                    CHECK(hipEventCreate(&stop_event));

                    CHECK(hipEventRecord(start_event, 0));
                    read_write<<<block_num, block_size>>>(A_d, B_d, elems);
                    CHECK(hipEventRecord(stop_event, 0));

                    CHECK(hipEventSynchronize(stop_event));

                    auto elapsed = float{};
                    CHECK(hipGetLastError());
                    CHECK(hipEventElapsedTime(&elapsed,
                                              start_event, stop_event));

                    mintime = std::min(mintime, elapsed);
                }

                file << "RW;" << block_size << ";" << block_num << ";"
                     << sizeof(float4) << ";" << elems << ";"
                     << mintime << ";"
                     << (2.0e-9 * sizeof(float4) * elems) / (mintime / 1e3)
                     << std::endl;
            }

            file << std::endl;

            for(auto block_num = sms;
                     block_num <= std::min(elems / block_size, max_blocks);
                     block_num *= 2)
            {
                // Initialize device memory
                CHECK(hipMemset(B_d, 0xff, sizeof(float4) * elems)); // NaN

                auto mintime = std::numeric_limits<float>::max();
                for(auto k = 0; k < iters; ++k)
                {
                    auto start_event = hipEvent_t{};
                    auto stop_event = hipEvent_t{};
                    CHECK(hipDeviceSynchronize());
                    CHECK(hipEventCreate(&start_event));
                    CHECK(hipEventCreate(&stop_event));

                    CHECK(hipEventRecord(start_event, 0));
                    write<<<block_num, block_size>>>(B_d, elems);
                    CHECK(hipEventRecord(stop_event, 0));

                    CHECK(hipEventSynchronize(stop_event));

                    auto elapsed = float{};
                    CHECK(hipGetLastError());
                    CHECK(hipEventElapsedTime(&elapsed, start_event,
                                               stop_event));

                    mintime = std::min(mintime, elapsed);
                }

                file << "WO;" << block_size << ";" << block_num << ";"
                     << sizeof(float4) << ";" << elems << ";"
                     << mintime << ";"
                     << (1.0e-9 * sizeof(float4) * elems) / (mintime / 1e3)
                     << std::endl;
            }

            file << std::endl;

            CHECK(hipFree(B_d));
            CHECK(hipFree(A_d));
        }
    }
}

auto main() -> int
{
    // set up devices
    auto dev_count = int{};
    CHECK(hipGetDeviceCount(&dev_count));

    std::cout << "Available accelerators: " << std::endl;
    for(auto i = 0; i < dev_count; ++i)
    {
        auto prop = hipDeviceProp_t{};
        CHECK(hipGetDeviceProperties(&prop, i));

        std::cout << "\t[" << i << "] " << prop.name << std::endl;
    }

    std::cout << std::endl;
    std::cout << "Select accelerator: ";
    auto index = 0;
    std::cin >> index;

    if(index >= dev_count)
    {
        std::cout << "I'm sorry, Dave. I'm afraid I can't do that."
                  << std::endl;
        return EXIT_FAILURE;
    }

    CHECK(hipSetDevice(index));
    CHECK(hipFree(nullptr));

    auto prop = hipDeviceProp_t{};
    CHECK(hipGetDeviceProperties(&prop, index));
    
    auto sms = prop.multiProcessorCount;
    auto max_blocks = prop.maxGridSize[0];

    auto now = std::chrono::system_clock::now();
    auto cnow = std::chrono::system_clock::to_time_t(now);

    auto filename = std::stringstream{};
    filename << "CUDA-";
    filename << std::put_time(std::localtime(&cnow), "%Y-%m-%d-%X");
    filename << ".csv";

    auto file = std::ofstream{filename.str()};

    file << "type;block_size;block_num;elem_size;elem_num;mintime;throughput"
         << std::endl;

    do_benchmark(sms, max_blocks, file, 64, 1024);

#ifdef KEPLER
    do_benchmark(sms, max_blocks, file, 192, 768);
#endif

    return EXIT_SUCCESS;
}
